#include "hip/hip_runtime.h"
//#include <ATen/native/SharedReduceOps.h> // WelfordData/WelfordOps
#include <ATen/native/cuda/Loops.cuh> // gpu kernel
#include <ATen/AccumulateType.h> // acc_type
#include <ATen/Tensor.h> // at::tensor
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/Dispatch.h> // at_dispatch macro
#include <c10/core/ScalarType.h>
#include <vector> // std::vector
#define MAX_THREADS_PER_BLOCK 512
#define MAX(a, b) (a > b) ? a : b
#define MIN(a, b) (a < b) ? a : b

template <typename T>
__device__ void
sum_reduce(
    T vals_reduced,
    const int start_stride,
    const int end_stride
  ) {
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
#pragma unroll 8
  for (int stride = start_stride; stride >= end_stride; stride >>= 1)
    if (tid < stride) {
      vals_reduced[tid] += vals_reduced[tid + stride];
    __syncthreads();
    }
}

template <typename T>
__global__ void
spatial_loop(
      const T* dy_data,
      const T* X_data,
      const int H,
      const int W,
      const int C,
      at::acc_type<T, true>* xdy_sum_data,
      at::acc_type<T, true>* dy_sum_data) {
  /*
     Performs a loop over the spatial dimension W, loading and summing dy and X. Spatial dimension H is processed in a separate kernel.
     C <= MAX_THREADS_PER_BLOCK (Kernel 1):
       griddim: (x=N, y=H, z=f=1); blockdim: (x=C, y=d)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        Cd = TPB (threads per block)
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, 1, C); X stride: (HWC, WC, dC, C, C, 1)
       shmem reduction: (d, C) -reduce-> C
       output buffer: (N, C, H)
     C > MAX_THREADS_PER_BLOCK (Kernel 2):
       griddim: (x=N, y=H, z=f); blockdim: (x=TPB, y=d=1)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        f * TPB = C
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, f, TPB); X stride: (HWC, WC, dC, C, TPB, 1)
       shmem reduction: (d, TPB) -reduce-> TPB
       output buffer: (N, f, TPB, H) -view-> (N, C, H)
   */

  using T_ACC = at::acc_type<T, true>;
  const int TPB = blockDim.y * blockDim.x;
  const int d = blockDim.y;
  const int w = W / d;
  T_ACC xdy_sum = 0;
  T_ACC dy_sum = 0;

#pragma unroll 8
  for (int i = 0; i < w; ++i) {
    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C; // dim 0, HWC stride
    reduce_idx += blockIdx.y * W * C; // dim 1, WC stride
    reduce_idx += i * d * C; // dim 2, dC stride
    reduce_idx += threadIdx.y * C; // dim 3, C stride
    reduce_idx += blockIdx.z * TPB; // dim 4, TPB stride (in kernel 1, threadIdx.z is always 0 so this statement does nothing)
    reduce_idx += threadIdx.x; // dim 5, 1 stride
    T_ACC dy_elem = static_cast<T_ACC>(dy_data[reduce_idx]);
    xdy_sum += dy_elem * X_data[reduce_idx];
    dy_sum += dy_elem;
  }

  // shmem reduction
  extern __shared__ char vals_reduced_uncasted[]; // size 2*TPB, TPB for sum1, TPB for sum2
  T *vals_reduced = reinterpret_cast<T*>(vals_reduced_uncasted);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  vals_reduced[2 * tid] = xdy_sum;
  vals_reduced[2 * tid + 1] = dy_sum;
  __syncthreads();
  sum_reduce(vals_reduced, TPB, 2 * C);

  // put reduced outputs into return buffers
  if (tid < C) {
    int out_idx = 0;
    out_idx += blockIdx.x * C * H; // dim 0, CH stride
    out_idx += blockIdx.z * TPB * H; // dim 1, TPB*H stride (if f=1, this line is a no-op)
    out_idx += threadIdx.x * H; // dim 2, H stride
    out_idx += blockIdx.y; // dim 3, 1 stride

    xdy_sum_data[out_idx] = vals_reduced[2 * tid];
    dy_sum_data[out_idx] = vals_reduced[2 * tid + 1];
  }
}

template <typename T>
__global__ void
compute_bwd_scale_biases(
    const T* mean_data,
    const T* rstd_data,
    const T* weight_data,
    at::acc_type<T, true>* xdy_sum_data,
    at::acc_type<T, true>* dy_sum_data,
    const int H,
    const int W,
    const int C,
    const int G,
    at::acc_type<T, true>* coef1_data,
    at::acc_type<T, true>* coef2_data,
    at::acc_type<T, true>* coef3_data) {
  /*
     griddim: (x=N); blockdim: (x=C)
      d = num. spatial elements (from HW dimension) each thread-block processes in parallel
      Cd = TPB (threads per block)
     X shape: (N, C) -view-> (N, G, D) -permute-> (N, D, G) -reduce-> (N, G)
     shmem reduction: (D, G) -reduce-> G
     output buffer: (N, G)
   */
  using T_ACC = at::acc_type<T, true>;
  const int D = C / G;
  const int n = blockIdx.x;
  const int c = threadIdx.x;
  const int g = c / D;
  const int d = c % D;
  const int nc = n * C + c;
  const T_ACC gamma_v = static_cast<T_ACC>(weight_data[c]);

  extern __shared__ char vals_reduced_uncasted[]; // size 2*C, C for sum1, C for sum2
  T_ACC *vals_reduced = reinterpret_cast<T_ACC*>(vals_reduced_uncasted);

  int idx = 0;
  idx += d * G;
  idx += g;
  vals_reduced[2 * idx] = xdy_sum_data[nc] * gamma_v;
  vals_reduced[2 * idx + 1] = dy_sum_data[nc] * gamma_v;
  __syncthreads();
  sum_reduce(vals_reduced, C, 2 * G);

  const int ng = n * G + g;
  const T_ACC mean_elem = static_cast<T_ACC>(mean_data[ng]);
  const T_ACC rstd_elem = static_cast<T_ACC>(rstd_data[ng]);
  coef1_data[nc] = rstd_elem * weight_data[c];

  if (d == 0) {
    const T_ACC sum1 = vals_reduced[2 * g];
    const T_ACC sum2 = vals_reduced[2 * g + 1];
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(D * H * W);
    const T_ACC x = (sum2 * mean_elem - sum1) * (rstd_elem * rstd_elem * rstd_elem * s);
    coef2_data[ng] = x;
    coef3_data[ng] = (-x * mean_elem) - (sum2 * rstd_elem * s);
  }
}

template <typename T>
__global__ void
compute_dweight_dbias(
    const T* mean_data,
    const T* rstd_data,
    at::acc_type<T, true>* xdy_sum_data,
    at::acc_type<T, true>* dy_sum_data,
    const int N,
    const int C,
    const int G,
    T* dweight_data,
    T* dbias_data) {
  // gridDim: (x=1), blockDim: (x=C)
  using T_ACC = at::acc_type<T, true>;
  const int c = threadIdx.x;
  const int D = C / G;
  const int g = c / D;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;

#pragma unroll 8
  for (int n = 0; n < N; ++n) {
    const int nc = n * C + c;
    const int ng = n * G + g;
    sum1 += ((xdy_sum_data[nc] - dy_sum_data[nc] * static_cast<T_ACC>(mean_data[ng])) * static_cast<T_ACC>(rstd_data[ng]));
    sum2 += dy_sum_data[nc];
  }
  dweight_data[c] = sum1;
  dbias_data[c] = sum2;
}

template <typename T>
void run_gn_bwd_kernels(
      const at::Tensor& dy_nhwc,
      const at::Tensor& X_nhwc,
      const at::Tensor& weight,
      const at::Tensor& mean,
      const at::Tensor& rstd,
      const int G,
      at::Tensor& dX,
      at::Tensor& dweight,
      at::Tensor& dbias
  ) {
  using T_ACC = at::acc_type<T, true>;
  const int N = X_nhwc.size(0);
  const int H = X_nhwc.size(1);
  const int W = X_nhwc.size(2);
  const int C = X_nhwc.size(3);
  const int D = C / G;

  const T* dy_data = dy_nhwc.const_data_ptr<T>();
  const T* X_data = X_nhwc.const_data_ptr<T>();
  const T* mean_data = mean.const_data_ptr<T>();
  const T* rstd_data = rstd.const_data_ptr<T>();
  const T* weight_data = weight.const_data_ptr<T>();

  const c10::ScalarType kAccType =
      (X_nhwc.scalar_type() == c10::ScalarType::Half || X_nhwc.scalar_type() == c10::ScalarType::BFloat16)
      ? at::kFloat
      : X_nhwc.scalar_type();

  at::Tensor xdy_dy_sum = at::empty({2, N, C, H}, X_nhwc.options().dtype(kAccType));
  T_ACC* xdy_sum_data = xdy_dy_sum.mutable_data_ptr<T_ACC>();
  T_ACC* dy_sum_data = xdy_sum_data + N * C * H;
  const int TPB = MIN(MAX_THREADS_PER_BLOCK, H * W * C);
  const int blockDimX = MIN(TPB, C);
  const int blockDimY = TPB / blockDimX;
  const int f = MAX(C / TPB, 1); // note: impossible for f > 1 AND blockDimY > 1
  spatial_loop<<<dim3(N, H, f), dim3(blockDimX, blockDimY), sizeof(T_ACC) * 2 * TPB>>>(
      dy_data, X_data, 
      H, W, C,
      xdy_sum_data, dy_sum_data);
  // sum over H dimension
  xdy_dy_sum = xdy_dy_sum.sum(3); // xdy_dy_sum shape now (2, N, C)
  xdy_sum_data = xdy_dy_sum.mutable_data_ptr<T_ACC>();
  dy_sum_data = xdy_sum_data + N * C;
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  T* dweight_data = dweight.mutable_data_ptr<T>();
  T* dbias_data = dbias.mutable_data_ptr<T>();
  compute_dweight_dbias<<<1, C>>>(
      mean_data, rstd_data,
      xdy_sum_data, dy_sum_data,
      N, C, G,
      dweight_data, dbias_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  at::Tensor coef1 = at::empty({N, C}, X_nhwc.options().dtype(kAccType));
  at::Tensor coef2 = at::empty({N, G}, X_nhwc.options().dtype(kAccType));
  at::Tensor coef3 = at::empty({N, G}, X_nhwc.options().dtype(kAccType));
  T_ACC* coef1_data = coef1.mutable_data_ptr<T_ACC>();
  T_ACC* coef2_data = coef2.mutable_data_ptr<T_ACC>();
  T_ACC* coef3_data = coef3.mutable_data_ptr<T_ACC>();
  compute_bwd_scale_biases<<<N, C, sizeof(T_ACC) * 2 * C>>>(
      mean_data, rstd_data, weight_data,
      xdy_sum_data, dy_sum_data,
      H, W, C, G,
      coef1_data, coef2_data, coef3_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  at::TensorIterator iter = at::TensorIteratorConfig()
                  .check_all_same_dtype(std::is_same<T, T_ACC>::value)
                  .resize_outputs(false)
                  .add_owned_output(dX.view({N, H * W, G, D}))
                  .add_owned_input(dy_nhwc.view({N, H * W, G, D}))
                  .add_owned_input(X_nhwc.view({N, H * W, G, D}))
                  .add_owned_input(coef1.view({N, 1, G, D}))
                  .add_owned_input(coef2.view({N, 1, G, 1}))
                  .add_owned_input(coef3.view({N, 1, G, 1}))
                  .build();
  at::native::gpu_kernel(
      iter, [] GPU_LAMBDA(T dy, T x, T_ACC coef1, T_ACC coef2, T_ACC coef3) -> T {
        return (coef1 * static_cast<T_ACC>(dy)) + (coef2 * static_cast<T_ACC>(x)) + coef3;
      });
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

std::vector<at::Tensor> gn_nhwc_cuda_bwd(
    const at::Tensor& dy,
    const at::Tensor& X,
    const at::Tensor& mean,
    const at::Tensor& rstd,
    const at::Tensor& weight,
    const int G
  ) {
  const int C = X.size(1);
  at::Tensor dy_nhwc = dy.permute({0, 2, 3, 1});
  at::Tensor X_nhwc = X.permute({0, 2, 3, 1});
  at::Tensor dX = at::empty_like(X_nhwc);
  at::Tensor dweight = at::empty({C}, X.options());
  at::Tensor dbias = at::empty({C}, X.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
    c10::ScalarType::Half,
    c10::ScalarType::BFloat16,
    X.scalar_type(),
    "group_norm_nhwc_backward", [&]() {
      run_gn_bwd_kernels<scalar_t>(
          dy_nhwc, X_nhwc,
          weight, mean, rstd,
          G,
          dX, dweight, dbias
      );
  });
  return {dX.permute({0, 3, 1, 2}), dweight, dbias};
}
