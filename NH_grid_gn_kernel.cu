#include "hip/hip_runtime.h"
//#include <ATen/native/SharedReduceOps.h> // WelfordData/WelfordOps
#include <ATen/AccumulateType.h> // acc_type
#include <ATen/ops/empty_like.h>
#include <ATen/Dispatch.h> // at_dispatch macro
#include <c10/core/ScalarType.h>
#include "scale_shift_kernel.h" // scale_shift
#include <thrust/pair.h> // thrust::pair
#include <vector> // std::vector
#define MAX_THREADS_PER_BLOCK 512 // 512 slightly faster (~3%) than 1024 because of higher theoretical occupancy -> higher mem throughput
#define MAX(a, b) (a > b) ? a : b
#define MIN(a, b) (a < b) ? a : b

template <typename scalar_t, typename index_t>
struct WelfordData {
  scalar_t mean;
  scalar_t m2;
  index_t n;
  scalar_t nf;

  C10_HOST_DEVICE WelfordData() : mean(0), m2(0), n(0), nf(0) {}

  C10_HOST_DEVICE WelfordData(
      scalar_t mean,
      scalar_t m2,
      index_t n,
      scalar_t nf)
      : mean(mean), m2(m2), n(n), nf(nf) {}
};


template <typename scalar_t, typename acc_scalar_t, typename index_t, typename res_t>
struct WelfordOps {
  acc_scalar_t correction;
  bool take_sqrt;
 public:
  using acc_t = WelfordData<acc_scalar_t, index_t>;
  inline C10_DEVICE acc_t reduce(acc_t acc, scalar_t data) const {
    // We accumulate n in index_t to avoid cumulative rounding error, but still
    // need nf for use in combine where int32 may overflow.
    index_t new_n = acc.n + 1;
    acc_scalar_t new_nf = static_cast<acc_scalar_t>(new_n);

    acc_scalar_t delta = data - acc.mean;
    
    acc_scalar_t new_mean = acc.mean + delta / new_nf;
    acc_scalar_t new_delta = data - new_mean;
    return {
      new_mean,
      acc.m2 + delta * new_delta,
      new_n,
      new_nf,
    };
  }
  inline C10_DEVICE acc_t combine(acc_t a, acc_t b) const {
    if (a.nf == 0) {
      return b;
    }
    if (b.nf == 0) {
      return a;
    }
    acc_scalar_t delta = b.mean - a.mean;
    acc_scalar_t new_count = a.nf + b.nf;
    acc_scalar_t nb_over_n = b.nf / new_count;
    return {
      a.mean + delta * nb_over_n,
      a.m2 + b.m2 + delta * delta * a.nf * nb_over_n,
      // setting acc.n as -1 since acc.n might not be able to represent the count
      // correctly within its range, setting it to -1 to avoid confusion
      -1,
      new_count
    };
  }
  inline C10_DEVICE res_t project(acc_t acc) const __ubsan_ignore_float_divide_by_zero__ {
    const auto mean = static_cast<scalar_t>(acc.mean);
    const auto divisor = acc.nf > correction ? acc.nf - correction : 0;
    const auto var = acc.m2 / divisor;
    res_t results(take_sqrt ? std::sqrt(var) : var, mean);
    return results;
  }

  static C10_DEVICE acc_t translate_idx(acc_t acc, int64_t /*base_idx*/) {
    return acc;
  }

#if defined(__HIPCC__) || defined(__HIPCC__)
  inline __device__ acc_t warp_shfl_down(acc_t acc, int offset) const {
    return {
      WARP_SHFL_DOWN(acc.mean, offset)
      , WARP_SHFL_DOWN(acc.m2, offset)
      , WARP_SHFL_DOWN(acc.n, offset)
      , WARP_SHFL_DOWN(acc.nf, offset)
    };
  }
#endif
  C10_HOST_DEVICE WelfordOps(acc_scalar_t correction, bool take_sqrt)
      : correction(correction), take_sqrt(take_sqrt) {}
};

template <typename T>
__global__ void
NH_compute_stats_pt1(
    const T* X,
    const int H,
    const int W,
    const int C,
    const int G,
    WelfordData<at::acc_type<T, true>, int> *welford_data
  ) {
  /*
     C <= MAX_THREADS_PER_BLOCK (Kernel 1):
       griddim: (x=N, y=H, z=f=1); blockdim: (x=C, y=d)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        Cd = TPB (threads per block)
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, f, C); X stride: (HWC, WC, dC, C, C, 1)
       shmem reduction: (d, C) -view-> (d, G, D) -permute-> (d, D, G) -reduce-> G
       output buffer: (N, 1, G, H)
     C > MAX_THREADS_PER_BLOCK (Kernel 2):
       griddim: (x=N, y=H, z=f); blockdim: (x=TPB, y=d=1)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        f * TPB = C
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, f, TPB); X stride: (HWC, WC, dC, C, TPB, 1)
       shmem reduction: (TPB,) -view-> (1, G/f, D) -permute-> (1, D, G/f) -reduce-> G/f
       output buffer: (N, f, G/f, H)
  */
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  const int TPB = blockDim.y * blockDim.x;
  const int d = blockDim.y;

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  const int w = W / d;
#pragma unroll 8
  for (int i = 0; i < w; ++i) {
    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C; // dim 0, HWC stride
    reduce_idx += blockIdx.y * W * C; // dim 1, WC stride
    reduce_idx += i * d * C; // dim 2, dC stride
    reduce_idx += threadIdx.y * C; // dim 3, C stride
    reduce_idx += blockIdx.z * TPB; // dim 4, TPB stride (in kernel 1, threadIdx.z is always 0 so this statement does nothing)
    reduce_idx += threadIdx.x; // dim 5, 1 stride
    T x = X[reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x)); // last arg isn't used in src
  }

  const int D = C / G;

  // shmem reduction
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int f = gridDim.z;
  const int gf = G / f;
  const int d_idx = threadIdx.y;
  const int gf_idx = threadIdx.x / D;
  const int D_idx = threadIdx.x % D;
  int idx = 0;
  idx += d_idx * D * gf; // dim 0, DG/f stride
  idx += D_idx * gf; // dim 1, G/f stride
  idx += gf_idx; // dim 2, 1 stride
  vals_reduced[idx] = val;
  __syncthreads();

  for (int stride = TPB / 2; stride >= gf; stride >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
    }

  // put reduced outputs into return buffers
  if (tid < gf) {
    int out_idx = 0;
    out_idx += blockIdx.x * G * H; // dim 0, HG stride
    out_idx += blockIdx.z * gf * H; // dim 2, G/f stride
    out_idx += threadIdx.x * H; // dim 3, 1 stride
    out_idx += blockIdx.y; // dim 1, G stride
    welford_data[out_idx] = vals_reduced[tid];
  }
}

template <typename T>
__global__ void
NH_compute_stats_pt2(
    WelfordData<at::acc_type<T, true>, int> *welford_data,
    const int H,
    const int G,
    const float eps,
    T* means,
    T* rstds
  ) {
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  /*
     griddim: (x=N, y=G); blockdim: (x=H)
      d = num. spatial elements (from H dimension) each thread-block processes in parallel
      Gd/f = TPB (threads per block)
     welford_data shape: (N, G, H); X stride: (GH, H, 1)
     shmem reduction: (H) -reduce-> 1
     output buffer: (N, G)
  */

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);
  const int TPB = blockDim.y * blockDim.x;

  // shmem reduction
  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  vals_reduced[tid] = welford_data[blockIdx.x * G * H + blockIdx.y * H + threadIdx.x];
  __syncthreads();

  for (int stride = TPB / 2; stride >= 1; stride >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
    }

  // put reduced outputs into return buffers
  if (tid < 1) {
    T_ACC m1, m2;
    thrust::tie(m2, m1) = welford_op.project(vals_reduced[tid]);
    int out_idx = 0;
    out_idx += blockIdx.x * G; // dim 0, G stride
    out_idx += blockIdx.y; // dim 1, G/f stride
    means[out_idx] = m1;
    rstds[out_idx] = rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
void NH_gn_fwd(
    const at::Tensor& X,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int G,
    T eps,
    at::Tensor& Y,
    at::Tensor& means,
    at::Tensor& rstds) {
  const T* X_data = X.const_data_ptr<T>();
  T* mean_data = means.mutable_data_ptr<T>();
  T* rstd_data = rstds.mutable_data_ptr<T>();

  const int N = X.size(0);
  const int H = X.size(1);
  const int W = X.size(2);
  const int C = X.size(3);

  //using WelfordType = at::native::WelfordData<at::acc_type<T, true>, int>;
  using WelfordType = WelfordData<at::acc_type<T, true>, int>;
  at::Tensor welford_tensor = at::empty({N, G, H, sizeof(WelfordType)}, X.options().dtype(at::kByte));
  WelfordType *welford_data = reinterpret_cast<WelfordType *>(welford_tensor.mutable_data_ptr());
  
  int blockDimX, blockDimY, f, TPB;
  TPB = MIN(MAX_THREADS_PER_BLOCK, W * C);

  blockDimX = MIN(TPB, C);
  blockDimY = TPB / blockDimX;
  f = MAX(C / TPB, 1); // note: impossible for f > 1 AND blockDimY > 1
  NH_compute_stats_pt1<<<dim3(N, H, f), dim3(blockDimX, blockDimY)>>>(
      X_data, H, W, C, G, 
      welford_data
  );

  TPB = MIN(MAX_THREADS_PER_BLOCK, H * G / f);
  blockDimX = MIN(TPB, G / f);
  blockDimY = TPB / blockDimX;
  NH_compute_stats_pt2<<<dim3(N, G), H>>>(
          welford_data,
          H, G, eps,
          mean_data, rstd_data
    );

  scale_shift<T>(X, weight, bias, G, Y, means, rstds);
  //AT_CUDA_CHECK(hipGetLastError());
}

std::vector<at::Tensor> gn_nhwc_cuda_fwd_NH_grid(
    const at::Tensor& X,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int G,
    float eps) {
  const int N = X.size(0);

  at::Tensor X_nhwc = X.permute({0, 2, 3, 1});
  at::Tensor X_out = at::empty_like(X_nhwc);
  at::Tensor means = at::empty({N, G}, weight.options());
  at::Tensor rstds = at::empty({N, G}, weight.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    X.scalar_type(),
    "group_norm_nhwc_forward_NH_grid", [&]() {
      NH_gn_fwd<scalar_t>(
          X_nhwc,
          weight, bias,
          G, eps,
          X_out, means, rstds
      );
  });
  return {X_out.permute({0, 3, 1, 2}), means, rstds};
}
