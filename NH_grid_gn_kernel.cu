#include "hip/hip_runtime.h"
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/cuda/Exceptions.h> // AT_CUDA_CHECK
#include <ATen/AccumulateType.h> // acc_type
#include <ATen/ops/empty_like.h>
#include <ATen/OpMathType.h> // opmath_t
#include <ATen/ops/empty.h>
#include <ATen/Dispatch.h> // at_dispatch macro
#include <ATen/Tensor.h> // torch tensor
#include <c10/cuda/CUDAMathCompat.h> // rsqrt
#include <c10/core/ScalarType.h>
#include <thrust/pair.h> // thrust::pair
#include <vector> // std::vector
#include "Welford.h"
#define MAX_THREADS_PER_BLOCK 512 // 512 slightly faster (~3%) than 1024 because of higher theoretical occupancy -> higher mem throughput
#define MAX(a, b) (a > b) ? a : b
#define MIN(a, b) (a < b) ? a : b

template <typename T>
__global__ void
compute_scale_biases(
        T* means,  // (N, G)
        T* rstds,  // (N, G)
        const T* weight, // (C)
        const T* bias,   // (C)
        const int G,
        const int C,
        at::acc_type<T, true>* a,            // (N, C)
        at::acc_type<T, true>* b             // (N, C)
  ) {
  // (N, f), (TPB)
  const int D = C / G;
  //const int c = threadIdx.x;
  const int c = blockIdx.y * blockDim.x + threadIdx.x;
  const int g = c / D;
  const int nc = blockIdx.x * C + c;
  const int ng = blockIdx.x * G + g;
  const at::acc_type<T, true> a_nc = rstds[ng] * weight[c];
  a[nc] = a_nc;
  b[nc] = -means[ng] * a_nc + bias[c];
}

template <typename T>
__device__ T
inline relu(T x) {
  return x > 0 ? x : 0;
}

template <typename T>
__device__ T
inline silu(T x) {
  using opmath_t = at::opmath_type<T>;
  return x / (opmath_t(1) + exp(-x));
}

template <typename T>
__device__ T
inline gelu(T x) {
  using opmath_t = at::opmath_type<T>;
  constexpr opmath_t kAlpha = M_SQRT1_2;
  return static_cast<opmath_t>(x) * opmath_t(0.5) * (opmath_t(1) + ::erf(static_cast<opmath_t>(x) * kAlpha));
}

template <typename T>
__device__ T
inline gelu_tanh(T x) {
  using opmath_t = at::opmath_type<T>;
  constexpr opmath_t kBeta = M_SQRT2 * M_2_SQRTPI * opmath_t(0.5);
  constexpr opmath_t kKappa = 0.044715;
  auto x_cube = static_cast<opmath_t>(x) * static_cast<opmath_t>(x) * static_cast<opmath_t>(x);
  auto inner = kBeta * (static_cast<opmath_t>(x) + kKappa * x_cube);
  return opmath_t(0.5) * static_cast<opmath_t>(x) * (opmath_t(1) + c10::cuda::compat::tanh(inner));
}

template <typename T, int num_elems>
struct float_vec;

template <typename T>
struct alignas(1 * sizeof(T)) float_vec<T, 1> {
  T x;
};
template <typename T>
struct alignas(2 * sizeof(T)) float_vec<T, 2> {
  T x, y;
};
template <typename T>
struct alignas(4 * sizeof(T)) float_vec<T, 4> {
  T x, y, z, w;
};

#define ACT 

template <typename T, int LOOP_I, int vec_elems>
__global__ void
scale_shift_elem_kernelV(
    const T* X_data,
    at::acc_type<T, true>* a_data,
    at::acc_type<T, true>* b_data,
    const int N,
    const int C,
    T* y
    ) {
  using T_ACC = at::acc_type<T, true>;
  using V = float_vec<T, vec_elems>;
  using V_ACC = float_vec<T_ACC, vec_elems>;
  const int n = (N * blockIdx.x) / gridDim.x;
  const int c = (blockIdx.y * blockDim.x + threadIdx.x) % (C / vec_elems);
  const int nc = n * (C / vec_elems) + c;
  const int num_vecs = gridDim.x * gridDim.y * LOOP_I * blockDim.x;
  const V *X_vec = reinterpret_cast<const V*>(X_data);
  V *y_vec = reinterpret_cast<V*>(y);
  V_ACC *a_vec = reinterpret_cast<V_ACC*>(a_data);
  V_ACC *b_vec = reinterpret_cast<V_ACC*>(b_data);
#pragma unroll LOOP_I
  for (int i = 0; i < LOOP_I; ++i) {
    int idx = 0;
    idx += blockIdx.x * LOOP_I * gridDim.y * blockDim.x;
    idx += i * gridDim.y * blockDim.x;
    idx += blockIdx.y * blockDim.x;
    idx += threadIdx.x;
    if (idx > num_vecs)
      continue;

    V tmp_X = X_vec[idx];
    V_ACC tmp_a = a_vec[nc];
    V_ACC tmp_b = b_vec[nc];
    if constexpr (vec_elems == 1)
      y_vec[idx] = {ACT(tmp_a.x * tmp_X.x + tmp_b.x)};
    else if constexpr (vec_elems == 2) {
      T y_x, y_y;
      y_x = ACT(tmp_a.x * tmp_X.x + tmp_b.x);
      y_y = ACT(tmp_a.y * tmp_X.y + tmp_b.y);
      y_vec[idx] = {y_x, y_y};
    }
    else if constexpr (vec_elems == 4) {
      T y_x, y_y, y_z, y_w;
      y_x = ACT(tmp_a.x * tmp_X.x + tmp_b.x);
      y_y = ACT(tmp_a.y * tmp_X.y + tmp_b.y);
      y_z = ACT(tmp_a.z * tmp_X.z + tmp_b.z);
      y_w = ACT(tmp_a.w * tmp_X.w + tmp_b.w);
      y_vec[idx] = {y_x, y_y, y_z, y_w};
    }
  }
}

template <typename T, int LOOP_I, int vec_elems>
__global__ void
small_scale_shift_elem_kernelV(
    const T* X_data,
    const T* mean_data,
    const T* rstd_data,
    const T* weight_data,
    const T* bias_data,
    const int N,
    const int C,
    const int G,
    T* y
    ) {
  using T_ACC = at::acc_type<T, true>;
  using V = float_vec<T, vec_elems>;
  const int n = (N * blockIdx.x) / gridDim.x;
  const int c = (blockIdx.y * blockDim.x + threadIdx.x) % (C / vec_elems);
  const int g = (G * c) / (C / vec_elems);
  const int ng = n * G + g;
  const V *x_vec = reinterpret_cast<const V*>(X_data);
  const V *weight_vec = reinterpret_cast<const V*>(weight_data);
  const V *bias_vec = reinterpret_cast<const V*>(bias_data);
  V *y_vec = reinterpret_cast<V*>(y);
  T mean = mean_data[ng];
  T rstd = rstd_data[ng];
  V weight_tmp = weight_vec[c];
  V bias_tmp = bias_vec[c];
#pragma unroll LOOP_I
  for (int i = 0; i < LOOP_I; ++i) {
    int idx = 0;
    idx += blockIdx.x * LOOP_I * gridDim.y * blockDim.x;
    idx += i * gridDim.y * blockDim.x;
    idx += blockIdx.y * blockDim.x;
    idx += threadIdx.x;

    V tmp_X = x_vec[idx];

    if constexpr (vec_elems == 1)
      y_vec[idx] = {ACT((static_cast<T_ACC>(tmp_X.x) - mean) * rstd * weight_tmp.x + bias_tmp.x)};
    else if constexpr (vec_elems == 2) {
      T y_x, y_y;
      y_x = ACT((static_cast<T_ACC>(tmp_X.x) - mean) * rstd * weight_tmp.x + bias_tmp.x);
      y_y = ACT((static_cast<T_ACC>(tmp_X.y) - mean) * rstd * weight_tmp.y + bias_tmp.y);
      y_vec[idx] = {y_x, y_y};
    }
    else if constexpr (vec_elems == 4) {
      T y_x, y_y, y_z, y_w;
      y_x = ACT((static_cast<T_ACC>(tmp_X.x) - mean) * rstd * weight_tmp.x + bias_tmp.x);
      y_y = ACT((static_cast<T_ACC>(tmp_X.y) - mean) * rstd * weight_tmp.y + bias_tmp.y);
      y_z = ACT((static_cast<T_ACC>(tmp_X.z) - mean) * rstd * weight_tmp.z + bias_tmp.z);
      y_w = ACT((static_cast<T_ACC>(tmp_X.w) - mean) * rstd * weight_tmp.w + bias_tmp.w);
      y_vec[idx] = {y_x, y_y, y_z, y_w};
    }
  }
}

template <typename T>
__global__ void
NH_compute_stats_pt1(
    const T* X,
    const int H,
    const int W,
    const int C,
    const int G,
    typename std::aligned_storage<4*sizeof(at::acc_type<T, true>), 4*sizeof(at::acc_type<T, true>)>::type *welford_data
  ) {
  /*
     C <= MAX_THREADS_PER_BLOCK (Kernel 1):
       griddim: (x=N, y=H, z=f=1); blockdim: (x=C, y=d)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        Cd = TPB (threads per block)
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, f, C); X stride: (HWC, WC, dC, C, C, 1)
       shmem reduction: (d, C) -view-> (d, G, D) -permute-> (d, D, G) -reduce-> G
       output buffer: (N, 1, G, H)
     C > MAX_THREADS_PER_BLOCK (Kernel 2):
       griddim: (x=N, y=H, z=f); blockdim: (x=TPB, y=d=1)
        f = factor of channels that each thread have to process separately
        d = num. spatial elements (from HW dimension) each thread-block processes in parallel
        f * TPB = C
       X shape: (N, H, W, C) -view-> (N, H, W/d, d, f, TPB); X stride: (HWC, WC, dC, C, TPB, 1)
       shmem reduction: (TPB,) -view-> (1, G/f, D) -permute-> (1, D, G/f) -reduce-> G/f
       output buffer: (N, f, G/f, H)
  */
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int>;
  using WelfordAligned = typename std::aligned_storage<4*sizeof(T_ACC), 4*sizeof(T_ACC)>::type;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  const int TPB = blockDim.y * blockDim.x;
  const int d = blockDim.y;

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  //const int w = W / d;
  const int w = ceil((float)W / d);
  int i;
#pragma unroll 8
  for (i = 0; i < w - 1; ++i) {
    //if ((int)(i * d + threadIdx.y) >= W)
    //  continue;

    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C; // dim 0, HWC stride
    reduce_idx += blockIdx.y * W * C; // dim 1, WC stride
    reduce_idx += i * d * C; // dim 2, dC stride
    reduce_idx += threadIdx.y * C; // dim 3, C stride
    reduce_idx += blockIdx.z * TPB; // dim 4, TPB stride (in kernel 1, threadIdx.z is always 0 so this statement does nothing)
    reduce_idx += threadIdx.x; // dim 5, 1 stride
    T x = X[reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x)); // last arg isn't used in src
  }
  if ((int)(i * d + threadIdx.y) < W) { // now i = w-1 and this condition isn't guaranteed to be true
    int reduce_idx = 0;
    reduce_idx += blockIdx.x * H * W * C; // dim 0, HWC stride
    reduce_idx += blockIdx.y * W * C; // dim 1, WC stride
    reduce_idx += i * d * C; // dim 2, dC stride
    reduce_idx += threadIdx.y * C; // dim 3, C stride
    reduce_idx += blockIdx.z * TPB; // dim 4, TPB stride (in kernel 1, threadIdx.z is always 0 so this statement does nothing)
    reduce_idx += threadIdx.x; // dim 5, 1 stride
    T x = X[reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x)); // last arg isn't used in src
  }

  const int D = C / G;

  // shmem reduction
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int f = gridDim.z;
  const int gf = G / f;
  const int d_idx = threadIdx.y;
  const int gf_idx = threadIdx.x / D;
  const int D_idx = threadIdx.x % D;

  int idx = 0;
  idx += d_idx * D * gf; // dim 0, DG/f stride
  idx += D_idx * gf; // dim 1, G/f stride
  idx += gf_idx; // dim 2, 1 stride
  vals_reduced[idx] = val;
  __syncthreads();

  int reduce_n = d * D;
#pragma unroll 8
  for (int stride = TPB / 2; stride >= gf && reduce_n % 2 == 0; stride >>= 1, reduce_n >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
    }

  if (tid < gf) {
    for (int di = 1; di < reduce_n; ++di)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + di*gf]);

    int out_idx = 0;
    out_idx += blockIdx.x * G * H; // dim 0, HG stride
    out_idx += blockIdx.z * gf * H; // dim 2, G/f stride
    out_idx += threadIdx.x * H; // dim 3, 1 stride
    out_idx += blockIdx.y; // dim 1, G stride
    welford_data[out_idx] = reinterpret_cast<WelfordAligned*>(&vals_reduced[tid])[0];
  }
}

template <typename T>
__global__ void
NH_compute_stats_pt2(
    typename std::aligned_storage<4*sizeof(at::acc_type<T, true>), 4*sizeof(at::acc_type<T, true>)>::type *welford_data,
    const int H,
    const int G,
    const float eps,
    T* means,
    T* rstds
  ) {
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  using WelfordAligned = typename std::aligned_storage<4*sizeof(T_ACC), 4*sizeof(T_ACC)>::type;
  /*
     griddim: (x=N, y=G); blockdim: (x=H)
      d = num. spatial elements (from H dimension) each thread-block processes in parallel
      Gd/f = TPB (threads per block)
     welford_data shape: (N, G, H); X stride: (GH, H, 1)
     shmem reduction: (H) -reduce-> 1
     output buffer: (N, G)
  */

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);
  const int TPB = blockDim.y * blockDim.x;

  // shmem reduction
  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[MAX_THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);
  WelfordAligned *vals_reduced_aligned = reinterpret_cast<WelfordAligned*>(vals_reduced_arr);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  //vals_reduced[tid] = reinterpret_cast<WelfordType*>(&welford_data[blockIdx.x * G * H + blockIdx.y * H + threadIdx.x])[0];
  vals_reduced_aligned[tid] = welford_data[blockIdx.x * G * H + blockIdx.y * H + threadIdx.x];
  __syncthreads();

  // next lowest power of 2 (AKA half of the next highest power of 2) - https://graphics.stanford.edu/%7Eseander/bithacks.html#RoundUpPowerOf2
  int start_stride = TPB - 1;
  start_stride |= start_stride >> 1;
  start_stride |= start_stride >> 2;
  start_stride |= start_stride >> 4;
  start_stride |= start_stride >> 8;
  start_stride |= start_stride >> 16;
  start_stride = (start_stride + 1) >> 1;

  // doing the first iteration outside the loop because of the extra condition regarding inputs with non-power-of-2 heights
  if (tid < start_stride && tid + start_stride < H)
    vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + start_stride]);
  __syncthreads();
#pragma unroll
  for (int stride = start_stride >> 1; stride >= 1; stride >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
    }

  // put reduced outputs into return buffers
  if (tid == 0) {
    T_ACC m1, m2;
    thrust::tie(m2, m1) = welford_op.project(vals_reduced[tid]);
    int out_idx = 0;
    out_idx += blockIdx.x * G; // dim 0, G stride
    out_idx += blockIdx.y; // dim 1, G/f stride
    means[out_idx] = m1;
    rstds[out_idx] = rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

#define TENSORIT 0

template <typename T>
void NH_gn_fwd(
    const at::Tensor& X,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int G,
    T eps,
    at::Tensor& Y,
    at::Tensor& means,
    at::Tensor& rstds) {
  const T* X_data = X.const_data_ptr<T>();
  T* mean_data = means.mutable_data_ptr<T>();
  T* rstd_data = rstds.mutable_data_ptr<T>();

  const int N = X.size(0);
  const int H = X.size(1);
  const int W = X.size(2);
  const int C = X.size(3);

  using T_ACC = at::acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int>;
  using WelfordAligned = typename std::aligned_storage<4*sizeof(T_ACC), 4*sizeof(T_ACC)>::type;
  at::Tensor welford_tensor = at::empty({N, G, H, sizeof(WelfordType)}, X.options().dtype(at::kByte));
  WelfordAligned *welford_data = reinterpret_cast<WelfordAligned *>(welford_tensor.mutable_data_ptr());
  
  int blockDimX, blockDimY, f, TPB;
  TPB = MIN(MAX_THREADS_PER_BLOCK, W * C);
  if (C < MAX_THREADS_PER_BLOCK)
    TPB -= TPB % C;
  else {
    int f = 1;
    while (C % f != 0 || C / f > MAX_THREADS_PER_BLOCK || G % f != 0) {
      f++;
    }
    TPB = C / f;
  }


  blockDimX = MIN(TPB, C);
  blockDimY = TPB / blockDimX;
  f = MAX(C / TPB, 1); // note: impossible for f > 1 AND blockDimY > 1
  NH_compute_stats_pt1<<<dim3(N, H, f), dim3(blockDimX, blockDimY)>>>(
      X_data, H, W, C, G, 
      welford_data
  );

  //printf("starting compute_stats_pt2 N: %d H %d W %d C %d G %d\n", N, H, W, C, G);
  NH_compute_stats_pt2<<<dim3(N, G), H>>>(
          welford_data,
          H, G, eps,
          mean_data, rstd_data
    );

  T* Y_data = Y.mutable_data_ptr<T>();

  if (H * W >= 1024) { // add fused scale-bias kernel to reduce num math ops on each element in the elementwise kernel if the spatial resolution is large
    const T* weight_data = weight.const_data_ptr<T>();
    const T* bias_data = bias.const_data_ptr<T>();

    const at::ScalarType kAccType =
        (X.scalar_type() == at::kHalf || X.scalar_type() == at::kBFloat16)
        ? at::kFloat
        : X.scalar_type();

    at::Tensor a = at::empty({N, C}, X.options().dtype(kAccType));
    at::Tensor b = at::empty({N, C}, X.options().dtype(kAccType));
    T_ACC* a_data = a.mutable_data_ptr<T_ACC>();
    T_ACC* b_data = b.mutable_data_ptr<T_ACC>();

    TPB = MIN(MAX_THREADS_PER_BLOCK, C);
    if (C < MAX_THREADS_PER_BLOCK)
      TPB -= TPB % C;
    else {
      int f = 1;
      while (C % f != 0 || C / f > MAX_THREADS_PER_BLOCK || G % f != 0) {
        f++;
      }
      TPB = C / f;
    }
    compute_scale_biases<<<dim3(N, f), TPB>>>( // note: max(D, T) threads per block
        mean_data, rstd_data,
        weight_data, bias_data,
        G, C,
        a_data, b_data);

    const int LOOP_I = 8;
    if (!TENSORIT && H * W * C % (C * LOOP_I) == 0) { // the modulus is somewhat arbitrary but ensures that the input is normal enough for the kernel to process correctly
      if (C % 4 == 0)
        scale_shift_elem_kernelV<T, LOOP_I, 4><<<dim3(N * H * W * C / TPB / LOOP_I / f / 4, f), TPB>>>(
            X_data,
            a_data, b_data,
            N, C,
            Y_data
            );
      else if (C % 2 == 0)
        scale_shift_elem_kernelV<T, LOOP_I, 2><<<dim3(N * H * W * C / TPB / LOOP_I / f / 2, f), TPB>>>(X_data, a_data, b_data, N, C, Y_data);
      else
        scale_shift_elem_kernelV<T, LOOP_I, 1><<<dim3(N * H * W * C / TPB / LOOP_I / f / 1, f), TPB>>>(X_data, a_data, b_data, N, C, Y_data);
    }
    else { 
      printf("using TensorIterator, N: %d H %d W %d C %d G %d TPB %d f %d\n", N, H, W, C, G, TPB, f);
      at::TensorIterator iter = at::TensorIteratorConfig()
        .check_all_same_dtype(std::is_same<T, T_ACC>::value) // this line relaxes requirement that all inputs/outputs are same dtype if T isn't T_ACC
        .resize_outputs(false)
        .add_owned_output(Y.view({N, H * W, C}))
        .add_owned_input(X.view({N, H * W, C}))
        .add_owned_input(a.view({N, 1, C}))
        .add_owned_input(b.view({N, 1, C}))
        .build();
     
      at::native::gpu_kernel(iter, [] GPU_LAMBDA(T x, T_ACC a, T_ACC b) -> T {
        return static_cast<T_ACC>(x) * a + b;
      });
    }
  }
  else { // if spatial resolution small, overhead of creating the extra kernel isn't worth it
    const int D = C / G;
    const int LOOP_I = 4;
    if (!TENSORIT && H * W * C % (C * LOOP_I) == 0) { // the modulus is somewhat arbitrary but ensures that the input is normal enough for the kernel to process correctly
      const T* weight_data = weight.const_data_ptr<T>();
      const T* bias_data = bias.const_data_ptr<T>();

      int vec_elems;
      if (D % 4 == 0) vec_elems = 4;
      else if (D % 2 == 0) vec_elems = 2;
      else vec_elems = 1;

      //printf("starting elem kernel N: %d H %d W %d C %d G %d vecelems %d TPB %d\n", N, H, W, C, G, vec_elems, TPB);
      if (vec_elems == 4)
        small_scale_shift_elem_kernelV<T, LOOP_I, 4><<<dim3(N * H * W * C / TPB / LOOP_I / f / vec_elems, f), TPB>>>(
            X_data,
            mean_data, rstd_data,
            weight_data, bias_data,
            N, C, G,
            Y_data
            );
      else if (vec_elems == 2)
        small_scale_shift_elem_kernelV<T, LOOP_I, 2><<<dim3(N * H * W * C / TPB / LOOP_I / f / vec_elems, f), TPB>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
      else
        small_scale_shift_elem_kernelV<T, LOOP_I, 1><<<dim3(N * H * W * C / TPB / LOOP_I / f / vec_elems, f), TPB>>>(X_data, mean_data, rstd_data, weight_data, bias_data, N, C, G, Y_data);
    }
    else {
      printf("using TensorIterator, N: %d H %d W %d C %d G %d TPB %d f %d\n", N, H, W, C, G, TPB, f);
      at::TensorIterator iter = at::TensorIteratorConfig()
        .check_all_same_dtype(std::is_same<T, T_ACC>::value) // this line relaxes requirement that all inputs/outputs are same dtype if T isn't T_ACC
        .resize_outputs(false)
        .add_owned_output(Y.view({N, H * W, G, D}))
        .add_owned_input(X.view({N, H * W, G, D}))
        .add_owned_input(means.view({N, 1, G, 1}))
        .add_owned_input(rstds.view({N, 1, G, 1}))
        .add_owned_input(weight.view({1, 1, G, D}))
        .add_owned_input(bias.view({1, 1, G, D}))
        .build();
       
      at::native::gpu_kernel(iter, [] GPU_LAMBDA(T x, T mean, T rstd, T weight, T bias) -> T {
        return (static_cast<T_ACC>(x) - mean) * rstd * weight + bias;
      });
    }
  }
  AT_CUDA_CHECK(hipGetLastError());
}

std::vector<at::Tensor> gn_nhwc_cuda_fwd_NH_grid(
    const at::Tensor& X,
    const at::Tensor& weight,
    const at::Tensor& bias,
    const int G,
    float eps) {
  const int N = X.size(0);

  at::Tensor X_nhwc = X.permute({0, 2, 3, 1});
  at::Tensor X_out = at::empty_like(X_nhwc);
  at::Tensor means = at::empty({N, G}, weight.options());
  at::Tensor rstds = at::empty({N, G}, weight.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    X.scalar_type(),
    "group_norm_nhwc_forward_NH_grid", [&]() {
      NH_gn_fwd<scalar_t>(
          X_nhwc,
          weight, bias,
          G, eps,
          X_out, means, rstds
      );
  });
  return {X_out.permute({0, 3, 1, 2}), means, rstds};
}
